#include <omp.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h> 
#include <float.h>
#include <hip/hip_runtime.h>


#define BLOCK_SIZE 512 // You can change this

__global__ void kernel(float *input, float *output, int len) {
	// Load a segment of the input vector into shared memory
	__shared__ float partialSum[2 * BLOCK_SIZE];
	unsigned int t = threadIdx.x, start = 2 * blockIdx.x * BLOCK_SIZE;
	if (start + t < len)
		partialSum[t] = input[start + t];
	else
		partialSum[t] = 0;
	if (start + BLOCK_SIZE + t < len)
		partialSum[BLOCK_SIZE + t] = input[start + BLOCK_SIZE + t];
	else
		partialSum[BLOCK_SIZE + t] = 0;
	// Traverse the reduction tree

	/*	
	   strides will assume values:
	   512
	   256
	   128
	   64
	   32
	   16
	   8
	   4
	   2
	   1
	*/ 
	for (unsigned int stride = BLOCK_SIZE; stride >= 1; stride >>= 1) {
		__syncthreads();
		if (t < stride)
			partialSum[t] += partialSum[t + stride];
	}
	// Write the computed sum of the block to the output vector at the
	// correct index
	if (t == 0)
		output[blockIdx.x] = partialSum[0];
}

float summation(float *input, int len) {
	float sum = 0.0;
	for (int i = 0; i < len; i++){
		sum += input[i];
	}
	return sum;
}
/*
void write_to_csv(double *T, int n_cells, const char *filename) {
         FILE *file = fopen(filename, "w");
         if (!file) {
                 fprintf(stderr, "Error: could not open file %s for writing. \n", filename);
                 return;
         }

         for (unsigned i = 0; i <= n_cells + 1; i++){
                 for (unsigned j = 0; j <= n_cells + 1; j++) {
                         fprintf(file, "%.6f", T(i,j));
                         if (j < n_cells + 1){
                                 fprintf(file, ",");
                         }
                 }
                 fprintf(file, "\n");
         }

         fclose(file);
         printf("Matrix saved to %s\n", filename);



 }
*/

int main(int argc, char *argv[]){
	
	//Generating 2D array of length 2^N (Passed from user input) filled with random floats. Floats are set to values such that the summation won't overflow the maximum value for float
	int N = 0;
	sscanf(argv[1], "%d", &N);
	
	int len = (int)pow(2, N);
	
	float *input = (float*)malloc(len * sizeof(float));

	if (input == NULL) {
    		printf("Memory allocation failed! Exiting.\n");
    		return -1;
	}


	srand(time(NULL));
	for (int i = 0; i < len; i++){
		input[i] = ((FLT_MAX / len) * 0.999999) * ((float) rand() / RAND_MAX);
		
	}
	
	//Print out the array to check that it was initialized and filled correctly
	for (int i = 0; i < len; i++){
		printf("%f \n", input[i]);
	}
	

	//Run the serial kernel and output the computation time in ms
	clock_t t;
	t = clock();

	//kernel call
	float sum_serial = summation(input, len);
	
	//Output sum to verify valid run
	printf("Sum: %f \n", sum_serial);

	t = clock() - t;
	printf("2^%d elements CPU Serial elapsed time: %f ms\n", N, ((double)t/CLOCKS_PER_SEC * 1000));


	int numBlocks = (len + 2 * BLOCK_SIZE - 1) / (2 * BLOCK_SIZE);
	int numThreads = BLOCK_SIZE;
	float *d_output;
	hipMalloc((void**)&d_output, sizeof(float));

	float *d_input;
	hipMalloc((void**)&d_input, len * sizeof(float));
	hipMemcpy(d_input, input, len * sizeof(float), hipMemcpyHostToDevice);


	hipEvent_t start, stop;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventRecord(start,0);

	// call device kernel
	kernel<<<numBlocks, numThreads>>>(d_input, d_output, len);

	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsedTime, start, stop);

	// Allocate memory for output sum on the host
	float output_CUDA = 0;

	// Copy result from GPU to CPU
	hipMemcpy(&output_CUDA, d_output, sizeof(float), hipMemcpyDeviceToHost);

	// Print the result
	printf("CUDA Sum: %f \n", output_CUDA);
	printf("2^%d elements CUDA elapsed time: %f ms\n", N, elapsedTime);
	
	//dim3 dimGrid((numOutputElements, 1, 1);
	//dim3 dimBlock((BLOCK_SIZE, 1, 1);
	//reduction<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, numInputElements);

}
